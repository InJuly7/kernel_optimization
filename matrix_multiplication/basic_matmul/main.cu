#include "hip/hip_runtime.h"
// This program computes a simple version of matrix multiplication
// By: Nick from CoffeeBeforeArch

#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>


// Matrix size of MxK  KxN  MxN;

__global__ void matrixMul(const float *a, const float *b, float *c) {
  // Compute each thread's global row and column index
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Iterate over row, and down column
  // c[row * N + col] = 0;
  float Cvalue = 0;
  for (int i = 0; i < K; i++) {
    // Accumulate results for a single element
    Cvalue += a[row * K + i] * b[i * N + col];
  }
  c[row * N + col] = Cvalue;
}

// Check result on the CPU
// MxN = MxK * KxN
 void verify_result(std::vector<float> &a, std::vector<float> &b, std::vector<float> &c, int M, int N, int K) {
   // For every row...
   for (int row = 0; row < M; row++) {
    // For every column...
     for (int col = 0; col < N; col++) {
      // For every element in the row-column pair
      float tmp = 0;
      for (int i = 0; i < K; i++) {
        // Accumulate the partial results
        tmp += a[row * N + i] * b[i * N + col];
      }
      // Check against the CPU result
      assert(tmp == c[row * N + col]);
     }
  }
}

int main() {

  int N = 1 << 10;
  // Size (in bytes) of matrix
  size_t Mat_bytes = N * N * sizeof(float);
  // Host vectors
  std::vector<float> h_a(M * K);
  std::vector<float> h_b(K * N);
  std::vector<float> h_c(M * N);

  // Initialize matrices
  std::generate(h_a.begin(), h_a.end(), []() { return rand() % 100; });
  std::generate(h_b.begin(), h_b.end(), []() { return rand() % 100; });

  // Allocate device memory
  float *d_a, *d_b, *d_c;
  hipMalloc(&d_a, MatA_bytes);
  hipMalloc(&d_b, MatB_bytes);
  hipMalloc(&d_c, MatC_bytes);

  // Copy data to the device
  hipMemcpy(d_a, h_a.data(), MatA_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b.data(), MatB_bytes, hipMemcpyHostToDevice);

  // Threads per CTA dimension
  int THREADS = 32;

  // Blocks per grid dimension (assumes THREADS divides N evenly)
  int BLOCK_X = M / THREADS;
  int BLOCK_Y = K / THREADS;
  // Use dim3 structs for block  and grid dimensions
  dim3 threads(THREADS, THREADS);
  dim3 blocks(BLOCK_X, BLOCK_Y);
	
  // Launch kernel
  matrixMul<<<blocks, threads>>>(d_a, d_b, d_c);

  // Copy back to the host
  hipMemcpy(h_c.data(), d_c, MatC_bytes, hipMemcpyDeviceToHost);

  // Check result
  // verify_result(h_a, h_b, h_c, M, N, K);
  std::cout << "COMPLETED SUCCESSFULLY\n";

  // Free memory on device
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}



